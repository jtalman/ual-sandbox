#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <gsl/gsl_randist.h>

#include "UAL/APDF/APDF_Builder.hh"
#include "PAC/Beam/Position.hh"
#include "SMF/PacSmf.h"
#include "PAC/Beam/Bunch.hh"
#include "Main/Teapot.h"
#include "UAL/UI/Shell.hh"
//#include "../../codes/common/book.h"
#include "SPINK/Propagator/DipoleTracker.hh"
#include "SPINK/Propagator/GpuTracker_hh.cu"
//#include "Def.cu"
//#include "TEAPOT/Integrator/RFCavityTracker.hh"
//#include "SPINK/Propagator/RFCavityTracker.hh"
#include "SPINK/Propagator/SpinTrackerWriter.hh"
//#include "SPINK/Propagator/SnakeTransform.hh"

#include "timer.h"
#include "PositionPrinter.h"
#include "SpinPrinter.h"

using namespace UAL;

int main(){

  UAL::Shell shell;

  double cc  = 2.99792458E+8;
  double G = 1.7928456;
  double mass   = 0.938272029;            //       proton mass [GeV]
  double charge = 1.0;

  /** AUL:17MAR10 _____________________________________________________________________*/
  /**********************************************************/
  //* Read input parameters*/
  /**********************************************************/
 
  std::ifstream configInput("./datagpu/spinkStep.in");//AULNLD:07JAN10

  std::string dummy ; // this string has been added to improve readability of input
  std::string variantName;
  bool outdmp;
  bool logdmp;
  int irSBend; int irQuad;
  double gamma;
  double tuneX, tuneY, chromX, chromY;
  bool settunes; //AUL:08MAR10
  double dgammadt; //double dgammadt = 1.1522754; // 1/s
  double V; double harmon ; //V = 1.5e-04, harmon = 0
  double ssx; double ssy; double ssz;
  double emit_y; // Pi mm*mrad (normalized) 15.; 
  double emit_x; // Pi mm*mrad (normalized)
  double emit_z; // Pi 
  double sigct, sigx,sigy,sigxp,sigyp,sigt;
  double x00,x00p,y00,y00p,dpp0,ct0; 
  int calcPhaseSpace;
  bool snkflag ; //AUL:10MAR10
  double mu1; double mu2; double phi1; double phi2; double the1; double the2;
  double stepsize;
  int turns, NPart, dumpstep;

  configInput >> dummy >> variantName;
  configInput >> dummy >> outdmp ; //AUL:12MAR10
  configInput >> dummy >> logdmp ; //std::cout << "logdmp=" << logdmp << std::endl;
  configInput >> dummy >> irSBend >> irQuad;
  configInput >> dummy >> gamma; 
  configInput >> dummy >> tuneX >> tuneY ;
  configInput >> dummy >> chromX >> chromY;
  configInput >> dummy >> settunes ; //AUL:08MAR10
  configInput >> dummy >> dgammadt;
  configInput >> dummy >> V >> harmon;
  configInput >> dummy >> ssx >> ssy >> ssz; 
  configInput >> dummy >> emit_x >> emit_y >> sigt;
  configInput >> dummy >> x00 >> x00p >> y00 >> y00p >> ct0 >> dpp0;
  configInput >> dummy >> calcPhaseSpace; 
  configInput >> dummy >> snkflag; 
  configInput >> dummy >> mu1 >> mu2 ; 
  configInput >> dummy >> phi1 >> phi2 ; 
  configInput >> dummy >> the1 >> the2 ;
  configInput >> dummy >> turns >> dumpstep;
  configInput >> dummy >> NPart;
  configInput >> dummy >> stepsize;

  std::cout << "emit =" << emit_x << " " << emit_y << " " << sigt << "\n";
  std::cout << "Dist =" << calcPhaseSpace << " snkflag = " << snkflag << " \n";
  std::cout << "mu = " << mu1 << " " << mu2 << " theta = " << the1 << " " << the2 << " turns = " << turns << " dumpstep =" << dumpstep << " NPart =" << NPart << " \n";
  std::cout << "stepsize = " << stepsize << " \n"; 
 // SPINK::SnakeTransform::setOutputDump(outdmp); //AUL:01MAR10
  SPINK::GpuTracker::setOutputDump(outdmp); //AUL:02MAR10
  // SPINK::RFCavityTracker::setOutputDump(outdmp); //AUL:27APR10
  // ************************************************************************
  if( logdmp ){std::cout << "\nDefine the space of Taylor maps." << std::endl;}
  // ************************************************************************

  shell.setMapAttributes(Args() << Arg("order", 5));

  // ************************************************************************
  if( logdmp ){  std::cout << "\nBuild lattice." << std::endl;}
  // ************************************************************************

  std::string sxfFile = "./datagpu/";
  sxfFile += variantName;
  sxfFile += ".sxf";

  std::cout << "sxfFile = " << sxfFile << endl;

  shell.readSXF(Args() << Arg("file",  sxfFile.c_str()));

  // ************************************************************************
  if( logdmp ){  std::cout << "\nAdd split ." << std::endl;}
  // ************************************************************************

  if( logdmp ){std::cout << "irSBend = " << irSBend << ", irQuad = " << irQuad << endl;}

  shell.addSplit(Args() << Arg("lattice", "rhic") << Arg("types", "Sbend")
  		 << Arg("ir", irSBend));

  shell.addSplit(Args() << Arg("lattice", "rhic") << Arg("types", "Quadrupole")
  		 << Arg("ir", irQuad));

  // ************************************************************************
  if( logdmp ){  std::cout << "Select lattice." << std::endl;}
  // ************************************************************************

  shell.use(Args() << Arg("lattice", "rhic"));

  // ************************************************************************
  if( logdmp ){  std::cout << "\nWrite ADXF file ." << std::endl;}
  // ************************************************************************

  std::string outputFile = "./outgpu/cpp/";
  outputFile += variantName;
  outputFile += ".sxf";

  shell.writeSXF(Args() << Arg("file",  outputFile.c_str()));

  // ************************************************************************
  if( logdmp ){std::cout << "\nDefine beam parameters." << std::endl;}
  // ************************************************************************

  double energy = gamma*mass;

  shell.setBeamAttributes(Args() << Arg("energy", energy) << Arg("mass", mass)
			  << Arg("charge",charge));

  PAC::BeamAttributes& ba = shell.getBeamAttributes();

  // ************************************************************************
  if( logdmp ){  std::cout << "\nLinear analysis." << std::endl;}
  // ************************************************************************
  
  // Make linear matrix

  std::string mapFile = "./outgpu/cpp/";
  mapFile += variantName;
  mapFile += ".map1";

  if( logdmp ){  std::cout << " matrix" << std::endl;}
  shell.map(Args() << Arg("order", 1) << Arg("print", mapFile.c_str()));

  // ************************************************************************
  if( logdmp ){  std::cout << "\nTune and chromaticity fitting. " << std::endl;}
  // ************************************************************************

  // shell.analysis(Args());

  /* for RHIC AUL:07MAY10 after a hint by Nikolay */
  if( settunes ){
    shell.tunefit(Args() << Arg("tunex", tuneX) << Arg("tuney", tuneY) << Arg("b1f", "^qf$") << Arg("b1d", "^qd$"));
      shell.chromfit(Args() << Arg("chromx", chromX) << Arg("chromy", chromY)<< Arg("b2f", "^sf") << Arg("b2d", "^sd"));
  } else {
    if( logdmp ){ std::cout << "\n--tunes and chromaticity NOT readjusted" << std::endl;}
  }
 
  
  /* for EDM AUL:07MAY10 after a hint by Nikolay
  if( settunes ){
    shell.tunefit(Args() << Arg("tunex", tuneX) << Arg("tuney", tuneY) << Arg("b1f", "^quadf$") << Arg("b1d", "^quadd$"));
    shell.chromfit(Args() << Arg("chromx", chromX) << Arg("chromy", chromY)<< Arg("b2f", "^sexf$") << Arg("b2d", "^sexd$"));
  } else {
    if( logdmp ){ std::cout << "\n--tunes and chromaticity NOT readjusted" << std::endl;}
  }
  */

  /* for SCT AUL:12MAY10 after a hint by Nikolay */
  // if( settunes ){
  // shell.tunefit(Args() << Arg("tunex", tuneX) << Arg("tuney", tuneY) << Arg("b1f", "^quada$") << Arg("b1d", "^quadb$"));
  // shell.chromfit(Args() << Arg("chromx", chromX) << Arg("chromy", chromY)<< Arg("b2f", "^sexta$") << Arg("b2d", "^sextb$"));
  // } else {
  // if( logdmp ){ std::cout << "\n--tunes and chromaticity NOT readjusted" << std::endl;}
  //}

  // Calculate twiss
  
  std::string twissFile = "./outgpu/cpp/";
  twissFile += variantName;
  twissFile += ".twiss";

  if( logdmp ){  std::cout << " twiss " << std::endl;}

  std::cout << "we are here \n";
  //shell.twiss(Args() << Arg("print", twissFile.c_str()));

  std::cout << "we are here next \n";
  std::cout << " calculate suml" << std::endl;
  shell.analysis(Args());

  SPINK::GpuTracker::setStep(stepsize);
  // ************************************************************************
  std::cout << "\nAlgorithm Part. " << std::endl;
  // ************************************************************************

  std::string apdfFile = "./datagpu/spink_gpu.apdf";

  UAL::APDF_Builder apBuilder;

  apBuilder.setBeamAttributes(ba);

  UAL::AcceleratorPropagator* ap = apBuilder.parse(apdfFile);

  if(ap == 0) {
    std::cout << "Accelerator Propagator has not been created " << std::endl;
    return 1;
  }
  if( logdmp ){
    std::cout << "\nSpink tracker, ";
    std::cout << "size : " << ap->getRootNode().size() << " propagators " << endl;

    // ************************************************************************
    std::cout << "\nSet Acceleration. " << std::endl;
    // ************************************************************************
  }

  // dgammadt = 2.094  V = 1.3e-4 
  double dedt = dgammadt*mass;
  double circum = OpticsCalculator::getInstance().suml; 
  double T_0 = circum / cc;

    double lag = asin((dedt * T_0)/(V))/(2*UAL::pi);
    double offset = lag*circum/harmon;
  // double lag = asin(dedt*T_0/V);
    // lag = 0.0;
    if( logdmp ){ }
    cout << "dgamma/dt = " << dgammadt << endl ; //AUL:29DEC09
    cout << "Circumference(m) = " << circum << endl;
    cout << "Volt = " << V << ", harmon =" << harmon << ", lag = " << lag << std::endl;
    double gamt = 24.5; // RHIC Transition gamma;
    double betak = sqrt(1.0 - 1.0/(gamma*gamma));
    double alpham = 1/(gamt*gamt); 
    double eta = alpham - 1.0/(gamma*gamma);
    double Qs0 = harmon*V*fabs(eta*cos(lag));
    Qs0 /= 2.0*UAL::pi*sqrt(betak)*energy;
    Qs0 = sqrt(Qs0);
    std::cout << "Qs0 = " << Qs0 <<  "\n";
    double sigdp = Qs0*(2.0*UAL::pi/T_0)*sigt/eta;
    std::cout << "sigdp = " << sigdp << "\n";
    std::cout << "eta =" << eta << " \n";
    std::cout << "sigt =" << sigt << "\n";
    emit_z = sigt*sigdp*UAL::pi;
    std::cout << "emit_z = " << emit_z << " \n";  
  

  SPINK::GpuTracker::setRF(V,harmon,lag);
  //  TEAPOT::RFCavityTracker  tracker;
  //tracker.setRF(V, harmon, lag);  //AUL:17MAR10
  //double circ = circum;
   SPINK::GpuTracker::setCircum(circum); //AUL:17MAR10



  // ************************************************************************
  if( logdmp ){  std::cout << "\nBunch Part." << std::endl;}
  // ************************************************************************

  ba.setG(G);         // proton G factor
  
  if( logdmp ){  cout << "gamma = " << gamma << ",  Ggamma = " << G*gamma << endl;}

 
  PAC::Bunch bunch(NPart);               // bunch with one particle
  bunch.setBeamAttributes(ba);

  if( logdmp ){  std::cout << "initial spin = " << ssx << "  " << ssy << "  " << ssz << std::endl;}

  PAC::Spin spin;
  spin.setSX(ssx);
  spin.setSY(ssy);
  spin.setSZ(ssz);

  //double amplit_y = 15.; // Pi mm*mrad (normalized) 15.; 
  //double amplit_x = 0.; // Pi mm*mrad (normalized)
  //double dpp0 = 0.0;
  
  double x0; double x0p; double y0; double y0p;

  if( logdmp ){
    std::cout << "emit_x = " << emit_x << ", emit_y = " << emit_y << std::endl; //AUL:30DEC09

    // ************************************************************************
    std::cout << "\nOptics" << std::endl; //AUL:30DEC09
    // ************************************************************************
  }
  UAL::OpticsCalculator& optics = UAL::OpticsCalculator::getInstance();

  optics.calculate();

  PacTwissData tws = optics.m_chrom->twiss();
  double q_x = tws.mu(0)/2./UAL::pi;
  double q_y = tws.mu(1)/2./UAL::pi;
  double beta_x = tws.beta(0);
  double beta_y = tws.beta(1);
  double chrm_x = optics.m_chrom->dmu(0)/2./UAL::pi;
  double chrm_y = optics.m_chrom->dmu(0)/2./UAL::pi;
  double alfa_x = tws.alpha(0);
  double alfa_y = tws.alpha(1);

 
    std::cout << "beta_x = " << beta_x << "  beta_y = " << beta_y << std::endl;
    std::cout << "alfa_x = " << alfa_x << "  alfa_y = " << alfa_y << std::endl;
    std::cout << "Q_x = " << q_x << "  Q_y = " << q_y << std::endl;
    std::cout << "chrom_x = " << chrm_x << "  chrom_y = " << chrm_y << std::endl;
   
  
    if(calcPhaseSpace >= 1){


      emit_x = emit_x*UAL::pi*1e-6; // /(gamma*6.0);
      emit_y = emit_y*UAL::pi*1e-6; // /(gamma*6.0);
    std::cout << "emit_x = " << emit_x << " emit_y = " << emit_y << " \n";
    
    
    double dp0 = 0.0;
    std::cout << "dp(0) = "<<tws.dp(0) << "dp(1) =" << tws.dp(1) << "\n";
    // ct0 = ct0 + offset*2;
    dp0 = sigdp;
    
 double gama_x = (1.0 + alfa_x*alfa_x)/beta_x;
 double gama_y = (1.0 + alfa_y*alfa_y)/beta_y;
 sigx  = sqrt(beta_x*emit_x/gamma + (tws.d(0)*sigdp)* (tws.d(0)*sigdp));
 sigxp = sigx*sqrt(gama_x/beta_x);
 sigy  = sqrt(beta_y*emit_y/gamma + (tws.d(1)*sigdp)*(tws.d(1)*sigdp));
 sigyp = sigy*sqrt(gama_y/beta_y);

 double corr_x = -alfa_x/sqrt(beta_x*gama_x);
 double corr_y = -alfa_y/sqrt(beta_x*gama_y);

 std::cout << "sigmas = " << sigx << " " << sigxp << " "<< sigy << " " << sigyp << " \n";
 std::cout << "corr_x = " << corr_x << "corr_y = " << corr_y << " \n";


 
 
  // index for phases, weights, and dp/p
 
  // number of phase angles psi between y and yprime
  int Npx,Npy,Npz,Nwx,Nwy,Nwz;
  Npx=Npy=8; Npz=0;
  // number of action weights for gaussian approximation
  int nw = 4;
  Nwx=Nwy=nw; Nwz=0;
  // weights for gaussian approximation
  double w[4] = { 0.2671, 0.94, 1.9617, 4.1589};
  double psi_x, psi_y, psi_z, J_x, J_y, J_z;    
 
  if(calcPhaseSpace == 1){

    std::cout << "Doing Random gaussian distribution \n";

  const gsl_rng_type *Tx, *Ty, *Ts;
  gsl_rng            *rx, *ry, *rs;
  double     rngx,rngxp,rngy,rngyp,rngs,rngdp;
  double bdry = 2.0;
  /* create a generator chosen by the environment variable GSL_RNG_TYPE */
  gsl_rng_env_setup();
  /* gsl_rng_default_seed = (long)getpid(); */
  gsl_rng_default_seed = 103;  Tx = gsl_rng_default;  rx = gsl_rng_alloc(Tx);
  //gsl_rng_set(rx, param->irandx);
  gsl_rng_default_seed = 10555;  Ty = gsl_rng_default;  ry = gsl_rng_alloc(Ty);
  //gsl_rng_set(ry, param->irandy);
  gsl_rng_default_seed = 1178;  Ts = gsl_rng_default;  rs = gsl_rng_alloc(Ts);
  
  for(int ip=0; ip < bunch.size(); ip ++ ){
 

  
    
do {
      gsl_ran_bivariate_gaussian(rx , 1.0, 1.0, corr_x, &rngx, &rngxp);
 } while ((rngx*rngx+rngxp*rngxp) > bdry);
    do { 
      gsl_ran_bivariate_gaussian(ry , 1.0, 1.0, corr_y, &rngy, &rngyp);
    } while( (rngy*rngy+rngyp*rngyp) > bdry);
    do {
      gsl_ran_bivariate_gaussian(rs , 1.0, 1.0, 0.0, &rngs , &rngdp);
    } while ( (rngs*rngs+rngdp*rngdp) > bdry);
    
  sigct = sigt*cc;
    x0  = rngx *sigx;
    x0p = rngxp*sigxp;
    y0  = rngy *sigy;
    y0p = rngyp*sigyp;
    ct0  = rngs *sigct + offset*2;
    dp0 = rngdp*sigdp;


 bunch[ip].getPosition().set(x0, x0p, y0, y0p, ct0, dp0);    //AUL:17MAR10
    bunch[ip].setSpin(spin);
     
    
      std::cout << ip << "  ";
      std::cout << " " << x0 << ",  " << x0p ;
     std::cout << " " << y0 << ", " << y0p ;
     std::cout << " " << ct0 << ", " << dp0 << std::endl;
   
  }
  
 // if(restart){
   // fclose (pFile);}
    gsl_rng_free (rx);
  gsl_rng_free (ry);
  gsl_rng_free (rs);} else { // Do discrete Gaussian distribution

    std::cout << "Doing Discrete Gaussian Distribution \n";

int ip = 0;

 std::cout << "Nwx = " << Nwx << " Nwy = " << Nwy << " Nwz =" << Nwz << " \n";
 std::cout << "Npx = " << Npx << " Npy = " << Npy << " Npz = " << Npz << " \n";

for(int iwx=0; iwx < Nwx; iwx ++){
    for(int iwy=0; iwy < Nwy; iwy ++){
       for(int ipx=0; ipx < Npx; ipx ++){
          for(int ipy=0; ipy < Npy; ipy++){
	    // for(int ipz=0; ipz < Npz; ipz++){
            //  for(int iwz=0;iwz < Nwz; iwz ++){
	    




   int	ipz = 0; int iwz = 0;
     psi_x = (2*UAL::pi/Npx)*ipx;
     psi_x = (2*UAL::pi/Npy)*ipy;
     psi_z = (2*UAL::pi/Npz)*ipz; 
     J_y = emit_y*exp(-w[iwy]/2)*0.5/(gamma);
     J_x = emit_x*exp(-w[iwx]/2)*0.5/(gamma);
     J_z = emit_z*exp(-w[iwz]/2)*0.5;

     x0 = sqrt(J_x*beta_x)*cos(psi_x);
     x0p = sqrt(J_x/beta_x)*(sin(psi_x) + alfa_x*cos(psi_x));
    

     y0 = sqrt(J_y*beta_y)*cos(psi_y);
     y0p = sqrt(J_y/beta_y)*(sin(psi_y) + alfa_y*cos(psi_y));
     
     ct0 =  offset*2;
     dp0 = 0.0;     

   bunch[ip].getPosition().set(x0, x0p, y0, y0p, ct0, dp0);    //AUL:17MAR10
    bunch[ip].setSpin(spin);
     
    
      std::cout << ip << "  ";
      std::cout << " " << x0 << ",  " << x0p ;
     std::cout << " " << y0 << ", " << y0p ;
     std::cout << " " << ct0 << ", " << dp0 << std::endl;
        ip++;
	    }}}}
//}}   
  
  }   

   // end of calcPhaseSpace 
    }else {
      std::cout << " starting loading of dist.in \n";
      precision Ggam,sx0,sy0,sz0,dp0;
      // else read in particle distribution from file
      std::ifstream distInput("dist.in");
      for(int ip=0; ip< bunch.size(); ip++){
	distInput >> gamma >> Ggam >> sx0 >> sy0 >> sz0 >> x0 >> x0p >> y0 >> y0p >> ct0 >> dp0;
	 std::cout << ip << "  ";
      std::cout << " " << x0 << ",  " << x0p ;
     std::cout << " " << y0 << ", " << y0p ;
     std::cout << " " << ct0 << ", " << dp0  <<  " ," << sx0 << ", " << sy0 << " ," << sz0 << std::endl;

       bunch[ip].getPosition().set(x0, x0p, y0, y0p, ct0, dp0);    //AUL:17MAR10
       //  bunch[ip].setSpin(spin);
    //  bunch[ip].getPosition().set(x0,x0p,y0,y0p,ct0,dp0);
	       spin.setSX(sx0);
		spin.setSY(sy0);
		spin.setSZ(sz0);
    	bunch[ip].setSpin(spin);
      }
      energy = gamma*mass;
  std::cout << "setting energy from file \n";
  
   shell.setBeamAttributes(Args() << Arg("energy", energy) << Arg("mass", mass)
  			  << Arg("charge",charge));
    std::cout << "after setting energy from file \n";
   ba = shell.getBeamAttributes();
   bunch.setBeamAttributes(ba);   

    }

  /** read in snake parameters AULNLD 2/9/10 */
 
 //SPINK::GpuDipoleTracker::loadPart(bunch);
 
  if( snkflag ){  //AUL:07MAY10

    SPINK::GpuTracker::setSnakeParams(mu1, mu2, phi1, phi2, the1, the2);

    if( logdmp ){    
      std::cout << "\nSnakes " << std::endl;
      std::cout << "snk1_mu = " << mu1 << ", snk2_mu = " << mu2 << endl;
      std::cout << "snk1_phi = " << phi1 << ", snk2_phi = " << phi2 << endl;
      std::cout << "snk1_theta = " << the1 << ", snk2_theta = " << the2 << endl;
    }
  }
  else 
  {
      if( logdmp ){
	std::cout << "\nNo Snakes" << std::endl ;
      }
  }

  // ************************************************************************
  if( logdmp ){  std::cout << "\nTracking. " << std::endl;}
  // ************************************************************************

  double t; // time variable
  
  //  if( logdmp ){ std::cout << "\nTurns = " << turns << std::endl ;}
  std::cout << "\nTurns = " << turns << std::endl ;
  //return 0;

  std::string orbitFile = "./outgpu/cpp/";
  orbitFile += variantName;
  orbitFile += ".orbit";

  PositionPrinter positionPrinter;
  positionPrinter.open(orbitFile.c_str());

  std::string spinFile = "./outgpu/cpp/";
  spinFile += variantName;
  spinFile += ".spin";
  
  SpinPrinter spinPrinter;
  spinPrinter.open(spinFile.c_str());
  int count;
  int step = 1;
  ba.setElapsedTime(0.0);

  start_ms();

  std::ofstream allpart, avgpart;

  //  allpart.open("PartOut.dat");
  avgpart.open("AvgOut.dat");
  char line[200];
  int N = bunch.size();
  precision Ggam = G*gamma;
  precision SxAvg =0.00, SyAvg=0.00, SzAvg=0.00;
  int stepdump = 0;
  
  SPINK::GpuTracker::setNturns(step);

 





 
   for(int iturn = 1; iturn <= turns; iturn++){

    /** to pass turn no for diagnostics AUL:02MAR10 */
    //  SPINK::SnakeTransform::setNturns(iturn);
  
    // SPINK::RFCavityTracker::setNturns(iturn);//AUL:27APR10
// for(int ip=0; ip < bunch.size(); ip++){
  //     positionPrinter.write(iturn, ip, bunch);
    //   spinPrinter.write(iturn, ip, bunch);
   // }

 
    
   SPINK::GpuTracker::GpuProp(bunch);

   // ap->propagate(bunch);
   //if( iturn % 10 == 0 ){
    avgpart << iturn*step << " ";
    // SPINK::GpuTracker::GpuPropagate(bunch);
    SPINK::GpuTracker::readPart(bunch,0);
     gamma = Energy[0]/mass;
     Ggam  = gamma*G; 
     SxAvg = 0.00; SyAvg=0.00; SzAvg=0.00;
     count = 0;
     // if(iturn == 5) { std::cout << "setting new step size to" << stepsize << "\n";
       
       // }

     //  if(iturn == 10) { SPINK::GpuTracker::setStep(stepsize*20);}
    //} 
         for(int ip=0; ip < bunch.size(); ip++){
         
   if(pos[ip].x*pos[ip].px*pos[ip].y*pos[ip].py*pos[ip].ct*pos[ip].de != pos[ip].x*pos[ip].px*pos[ip].y*pos[ip].py*pos[ip].ct*pos[ip].de ){ 
     }else {count++;
     SxAvg += pos[ip].sx; SyAvg += pos[ip].sy; SzAvg += pos[ip].sz;

     }
       }
	 int ip = 0;
	 sprintf(line," %i  %e  %e  %e  %e  %e  %e  %e  %e  %e  %e  %e \n",count,gamma,Ggam,SxAvg/count,SyAvg/count,SzAvg/count,pos[ip].x,pos[ip].px,pos[ip].y,pos[ip].py,pos[ip].ct,pos[ip].de);
	 avgpart << line ;


	 if(iturn % dumpstep == 0) {
           std::string PartFile = "PartOut";
           char dumpNo[3];
	   sprintf(dumpNo, "%d",stepdump);
	   stepdump++;
           PartFile += dumpNo;
           PartFile += ".dat";
           const char * inputc = PartFile.c_str();

           allpart.open(inputc);
for(int ip = 0; ip < N; ip++) {
    	 sprintf(line," %e  %e  %e  %e  %e  %e  %e  %e  %e  %e  %e \n",gamma,Ggam,pos[ip].sx,pos[ip].sy,pos[ip].sz,pos[ip].x,pos[ip].px,pos[ip].y,pos[ip].py,pos[ip].ct,pos[ip].de);
	 allpart << line ;
     }

   allpart.close();


	 }




  // }

    }
   // SPINK::GpuTracker::readPart(bunch,1);



     allpart.open("PartOut_last.dat");

   for(int ip = 0; ip < N; ip++) {
    	 sprintf(line," %e  %e  %e  %e  %e  %e  %e  %e  %e  %e  %e \n",gamma,Ggam,pos[ip].sx,pos[ip].sy,pos[ip].sz,pos[ip].x,pos[ip].px,pos[ip].y,pos[ip].py,pos[ip].ct,pos[ip].de);
	 allpart << line ;
     }

   allpart.close();
   avgpart.close();
  t = (end_ms());
  std::cout << "time  = " << t << " ms" << endl;
  positionPrinter.close();
  spinPrinter.close();
  //  hipDeviceReset();

  return 1;
}


